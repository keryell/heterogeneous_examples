/*
  @author Ken O'Brien <kennetho@xilinx.com>

  Parallel vector addition for CUDA devices.
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

constexpr size_t N = 3;
using Vector = float[N];


__global__ void vector_add(const float *a, const float *b, float *c) {
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if(idx < N)
    c[idx] = a[idx] + b[idx];
}

void checkError(hipError_t err) {
  if(err != hipSuccess) {
    throw std::domain_error("CUDA ERROR: "
     + std::string { hipGetErrorString(err) });
  }
}


int main() {
  Vector a = {1, 2, 3};
  Vector b = {5, 6, 8};
  Vector c;

  float *d_a, *d_b, *d_c;

  checkError(hipMalloc((void**) &d_a, N*sizeof(*d_a)));
  checkError(hipMalloc((void**) &d_b, N*sizeof(*d_b)));
  checkError(hipMalloc((void**) &d_c, N*sizeof(*d_c)));

  checkError(hipMemcpy(d_a, a, N*sizeof(*a), hipMemcpyHostToDevice));
  checkError(hipMemcpy(d_b, b, N*sizeof(*b), hipMemcpyHostToDevice));

  vector_add<<<1, N>>>(d_a, d_b, d_c);

  checkError(hipMemcpy(c, d_c, N*sizeof(*c), hipMemcpyDeviceToHost));

  std::cout << std::endl << "Result: " << std::endl;
  for(auto e: c)
    std::cout << e << " ";
  std::cout << std::endl;

  checkError(hipFree(d_a));
  checkError(hipFree(d_b));
  checkError(hipFree(d_c));

  return 0;
}
